
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#define N 1024

__global__ void Matriz_GPU_Mult(int *a, int *b, int *c) {
	int k, sum = 0;
	int i = blockIdx.x * blockDim.x + threadIdx.x; 
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  if (i < N && j < N) {
    for (k = 0; k < N; k++) {
      sum += a[j * N + k] * b[k * N + i];
    }
    c[j * N + i] = sum;
  }
}

int main() {
  double timeGPU;
  int A[N][N], B[N][N], C[N][N];
  int *d_a, *d_b, *d_c;
  int cont,i,j;

  //inicializacion
	for (i = 0; i < N; i++) {
  	cont = 0;
  	for (j = 0; j < N; j++) {
   		A[i][j] = cont;
   		B[i][j] = cont;
   		cont++;
  	}
  }

  int size = N * N * sizeof(int);
  hipMalloc((void **) &d_a, size);
  hipMalloc((void **) &d_b, size);
  hipMalloc((void **) &d_c, size);

  hipMemcpy(d_a, A, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, B, size, hipMemcpyHostToDevice);

  //int threadsPerBlock(16);
  //int numBlocks(N/threadsPerBlock);
  dim3 threadsPerBlock(32, 32);
  dim3 numBlocks(N/threadsPerBlock.x, N/threadsPerBlock.y);
  clock_t startGPU  = clock();
  Matriz_GPU_Mult<<<numBlocks, threadsPerBlock>>>(d_a, d_b, d_c);
	timeGPU = ((double)(clock() - startGPU))/CLOCKS_PER_SEC;
  
  hipMemcpy(C, d_c, size, hipMemcpyDeviceToHost);
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
  // tiempos de ejecucion
  printf("tiempo GPU = %f s",timeGPU);
  return 0;
}
