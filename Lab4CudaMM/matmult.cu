#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>
#define N 1024

__global__ void Matriz_GPU_Mult(int *a, int *b, int *c) {
	int k, sum = 0;
	int i = blockIdx.x * blockDim.x + threadIdx.x; 
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  if (i < N && j < N) {
    for (k = 0; k < N; k++) {
      sum += a[j * N + k] * b[k * N + i];
    }
    c[j * N + i] = sum;
  }
}

int main() {
  double timeGPU;
  int h_A[N][N], h_B[N][N], h_C[N][N];
  int *d_a, *d_b, *d_c;
  
  hipMalloc((void **) &d_a, N*sizeof(int));
  hipMalloc((void **) &d_b, N*sizeof(int));
  hipMalloc((void **) &d_c, N*sizeof(int));

  hipMemcpy(d_a, A, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, B, size, hipMemcpyHostToDevice);

  //int threadsPerBlock(16);
  //int numBlocks(N/threadsPerBlock);
  dim3 threadsPerBlock(32, 32);
  dim3 numBlocks(N/threadsPerBlock.x, N/threadsPerBlock.y);
  clock_t startGPU  = clock();
  Matriz_GPU_Mult<<<numBlocks, threadsPerBlock>>>(d_a, d_b, d_c);
	timeGPU = ((double)(clock() - startGPU))/CLOCKS_PER_SEC;
  hipMemcpy(C, d_c, size, hipMemcpyDeviceToHost);
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
  // tiempos de ejecucion
  printf("tiempo GPU = %f s",timeGPU);
  return 0;
}
